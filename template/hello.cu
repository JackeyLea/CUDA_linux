
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world_kernel(){
    printf("Hello World");
}

int main(){
    hello_world_kernel<<<1,1>>>();
    hipDeviceSynchronize();
}
