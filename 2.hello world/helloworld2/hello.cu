#include "hip/hip_runtime.h"
#include "hello.h"

extern "C"
__global__ void hellofromGPU(void)
{
    printf("GPU:hello sunyi\n");
}

void showhello(void)
{
    hellofromGPU <<<1,10>>>();
    hipDeviceSynchronize();
}
